#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <vector>
#include <algorithm>
#include <thrust/device_vector.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <glm/glm.hpp>
#include <glm/gtx/component_wise.hpp>
#include <glm/gtc/constants.hpp>
#include <glm/gtc/type_ptr.hpp>

// Compile with
// nvcc -Xcompiler -fPIC -shared -o raycast_volume.so raycast_volume.cu

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}
extern "C"
__host__ __device__
void printVec(const char *str, glm::vec3 vec) {
  printf("%s: Vec3(%.2f, %.2f, %.2f)\n", str, vec.x, vec.y, vec.z);
}

__host__ __device__ float3 vec2float3(glm::vec3 v) {
  return make_float3(v.x, v.y, v.z);
}

texture<float, hipTextureType3D, hipReadModeElementType> tex;
texture<float, hipTextureType1D, hipReadModeElementType> tf;

__device__ inline float texFetchVolume(glm::vec3 p, bool useTF=false) {
  if (useTF) return tex1D(tf, tex3D(tex, p.x, p.y, p.z));
  else       return           tex3D(tex, p.x, p.y, p.z);
}

__device__ inline float texFetchVolumeWorld(glm::vec3 worldPos,
                                            glm::vec3 worldBounds, bool useTF=false) {
  glm::vec3 idx = worldPos / worldBounds;
  if (useTF) return tex1D(tf, tex3D(tex, idx.x, idx.y, idx.z));
  else       return           tex3D(tex, idx.x, idx.y, idx.z);
}

__device__ inline float texFetchVolumeWorld(float x, float y, float z, glm::vec3 worldBounds, bool useTF=false) {
  if (useTF) return tex1D(tf, tex3D(tex, x / worldBounds.x, y / worldBounds.y, z / worldBounds.z));
  else       return           tex3D(tex, x / worldBounds.x, y / worldBounds.y, z / worldBounds.z);
}

__device__
bool hasVisibleNeighbors(glm::vec3 pos, glm::vec3 volSize, bool useTF=true) {
    glm::vec3 offset = 1.f / volSize;
    float transparencySum = (
        texFetchVolume(pos + offset * glm::vec3(-1.f, -1.f, -1.f), useTF) +
        texFetchVolume(pos + offset * glm::vec3(-1.f, -1.f, 0.f),  useTF) +
        texFetchVolume(pos + offset * glm::vec3(-1.f, -1.f, 1.f),  useTF) +
        texFetchVolume(pos + offset * glm::vec3(-1.f, 0.f, -1.f),  useTF) +
        texFetchVolume(pos + offset * glm::vec3(-1.f, 0.f, 0.f),   useTF) +
        texFetchVolume(pos + offset * glm::vec3(-1.f, 0.f, 1.f),   useTF) +
        texFetchVolume(pos + offset * glm::vec3(-1.f, 1.f, -1.f),  useTF) +
        texFetchVolume(pos + offset * glm::vec3(-1.f, 1.f, 0.f),   useTF) +
        texFetchVolume(pos + offset * glm::vec3(-1.f, 1.f, 1.f),   useTF) +
        texFetchVolume(pos + offset * glm::vec3(0.f, -1.f, -1.f),  useTF) +
        texFetchVolume(pos + offset * glm::vec3(0.f, -1.f, 0.f),   useTF) +
        texFetchVolume(pos + offset * glm::vec3(0.f, -1.f, 1.f),   useTF) +
        texFetchVolume(pos + offset * glm::vec3(0.f, 0.f, -1.f),   useTF) +
        texFetchVolume(pos + offset * glm::vec3(0.f, 0.f, 0.f),    useTF) +
        texFetchVolume(pos + offset * glm::vec3(0.f, 0.f, 1.f),    useTF) +
        texFetchVolume(pos + offset * glm::vec3(0.f, 1.f, -1.f),   useTF) +
        texFetchVolume(pos + offset * glm::vec3(0.f, 1.f, 0.f),    useTF) +
        texFetchVolume(pos + offset * glm::vec3(0.f, 1.f, 1.f),    useTF) +
        texFetchVolume(pos + offset * glm::vec3(1.f, -1.f, -1.f),  useTF) +
        texFetchVolume(pos + offset * glm::vec3(1.f, -1.f, 0.f),   useTF) +
        texFetchVolume(pos + offset * glm::vec3(1.f, -1.f, 1.f),   useTF) +
        texFetchVolume(pos + offset * glm::vec3(1.f, 0.f, -1.f),   useTF) +
        texFetchVolume(pos + offset * glm::vec3(1.f, 0.f, 0.f),    useTF) +
        texFetchVolume(pos + offset * glm::vec3(1.f, 0.f, 1.f),    useTF) +
        texFetchVolume(pos + offset * glm::vec3(1.f, 1.f, -1.f),   useTF) +
        texFetchVolume(pos + offset * glm::vec3(1.f, 1.f, 0.f),    useTF) +
        texFetchVolume(pos + offset * glm::vec3(1.f, 1.f, 1.f),    useTF)
    );
    return transparencySum > 0.f;
}

__device__ glm::vec3 computeGradientSobel(glm::vec3 pos, glm::ivec3 volSize,
                                          glm::vec3 voxelScale) {
  glm::vec3 offset = 0.1f / (glm::vec3(volSize) * voxelScale);
  float gx = (texFetchVolume(pos + offset * glm::vec3(1.f, 0.f, 0.f)) * 4.f +
              texFetchVolume(pos + offset * glm::vec3(1.f, 0.f, 1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(1.f, 1.f, 0.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(1.f, 0.f, -1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(1.f, -1.f, 0.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(1.f, 1.f, 1.f)) +
              texFetchVolume(pos + offset * glm::vec3(1.f, 1.f, -1.f)) +
              texFetchVolume(pos + offset * glm::vec3(1.f, -1.f, 1.f)) +
              texFetchVolume(pos + offset * glm::vec3(1.f, -1.f, -1.f))) -
             (texFetchVolume(pos + offset * glm::vec3(-1.f, 0.f, 0.f)) * 4.f +
              texFetchVolume(pos + offset * glm::vec3(-1.f, 0.f, 1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(-1.f, 1.f, 0.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(-1.f, 0.f, -1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(-1.f, -1.f, 0.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(-1.f, 1.f, 1.f)) +
              texFetchVolume(pos + offset * glm::vec3(-1.f, 1.f, -1.f)) +
              texFetchVolume(pos + offset * glm::vec3(-1.f, -1.f, 1.f)) +
              texFetchVolume(pos + offset * glm::vec3(-1.f, -1.f, -1.f)));

  float gy = (texFetchVolume(pos + offset * glm::vec3(0.f, 1.f, 0.f)) * 4.f +
              texFetchVolume(pos + offset * glm::vec3(0.f, 1.f, 1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(1.f, 1.f, 0.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(0.f, 1.f, -1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(-1.f, 1.f, 0.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(1.f, 1.f, 1.f)) +
              texFetchVolume(pos + offset * glm::vec3(1.f, 1.f, -1.f)) +
              texFetchVolume(pos + offset * glm::vec3(-1.f, 1.f, 1.f)) +
              texFetchVolume(pos + offset * glm::vec3(-1.f, 1.f, -1.f))) -
             (texFetchVolume(pos + offset * glm::vec3(0.f, -1.f, 0.f)) * 4.f +
              texFetchVolume(pos + offset * glm::vec3(0.f, -1.f, 1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(1.f, -1.f, 0.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(0.f, -1.f, -1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(-1.f, -1.f, 0.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(1.f, -1.f, 1.f)) +
              texFetchVolume(pos + offset * glm::vec3(1.f, -1.f, -1.f)) +
              texFetchVolume(pos + offset * glm::vec3(-1.f, -1.f, 1.f)) +
              texFetchVolume(pos + offset * glm::vec3(-1.f, -1.f, -1.f)));

  float gz = (texFetchVolume(pos + offset * glm::vec3(0.f, 0.f, 1.f)) * 4.f +
              texFetchVolume(pos + offset * glm::vec3(0.f, 1.f, 1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(1.f, 0.f, 1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(0.f, -1.f, 1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(-1.f, 0.f, 1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(1.f, 1.f, 1.f)) +
              texFetchVolume(pos + offset * glm::vec3(1.f, -1.f, 1.f)) +
              texFetchVolume(pos + offset * glm::vec3(-1.f, 1.f, 1.f)) +
              texFetchVolume(pos + offset * glm::vec3(-1.f, -1.f, 1.f))) -
             (texFetchVolume(pos + offset * glm::vec3(0.f, 0.f, -1.f)) * 4.f +
              texFetchVolume(pos + offset * glm::vec3(0.f, 1.f, -1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(1.f, 0.f, -1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(0.f, -1.f, -1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(-1.f, 0.f, -1.f)) * 2.f +
              texFetchVolume(pos + offset * glm::vec3(1.f, 1.f, -1.f)) +
              texFetchVolume(pos + offset * glm::vec3(1.f, -1.f, -1.f)) +
              texFetchVolume(pos + offset * glm::vec3(-1.f, 1.f, -1.f)) +
              texFetchVolume(pos + offset * glm::vec3(-1.f, -1.f, -1.f)));

  return glm::vec3(gx, gy, gz);
}

__device__ glm::vec3 computeGradient(glm::vec3 worldPos, glm::vec3 worldBounds) {
  // glm::vec3 offset = 1.0f;// / worldBounds;
  float gx =
      texFetchVolumeWorld(worldPos.x + 1.0f, worldPos.y, worldPos.z, worldBounds) -
      texFetchVolumeWorld(worldPos.x - 1.0f, worldPos.y, worldPos.z, worldBounds);
  float gy =
      texFetchVolumeWorld(worldPos.x, worldPos.y + 1.0f, worldPos.z, worldBounds) -
      texFetchVolumeWorld(worldPos.x, worldPos.y - 1.0f, worldPos.z, worldBounds);
  float gz =
      texFetchVolumeWorld(worldPos.x, worldPos.y, worldPos.z + 1.0f, worldBounds) -
      texFetchVolumeWorld(worldPos.x, worldPos.y, worldPos.z - 1.0f, worldBounds);
  return glm::vec3(gx, gy, gz);
}

extern "C" hipArray *uploadVolume(float *vol, glm::ivec3 volSizeVec) {
  hipExtent volSize =
      make_hipExtent(volSizeVec.x, volSizeVec.y, volSizeVec.z);
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

  hipArray *cuArray;
  gpuErrchk(hipMalloc3DArray(&cuArray, &channelDesc, volSize));
  hipMemcpy3DParms cpyParams = {0};
  cpyParams.srcPtr =
      make_hipPitchedPtr((void *)vol, volSize.width * sizeof(float),
                          volSize.width, volSize.height);
  cpyParams.dstArray = cuArray;
  cpyParams.extent = volSize;
  cpyParams.kind = hipMemcpyHostToDevice;
  gpuErrchk(hipMemcpy3D(&cpyParams));

  tex.normalized = true;
  tex.filterMode = hipFilterModeLinear;
  tex.addressMode[0] = hipAddressModeBorder;
  tex.addressMode[1] = hipAddressModeBorder;
  tex.addressMode[2] = hipAddressModeBorder;
  gpuErrchk(hipBindTextureToArray(tex, cuArray, channelDesc));

  return cuArray;
}

extern "C" hipArray *uploadTransferFunction(float *tf_p,
                                             size_t tf_resolution) {
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

  // float* d_tf;
  hipArray *d_tf;
  // size_t offset = 0;
  // gpuErrchk(hipMalloc((void**) &d_tf, tf_resolution * sizeof(float)));
  gpuErrchk(hipMallocArray(&d_tf, &channelDesc, tf_resolution));
  // gpuErrchk(hipMemcpy(d_tf, tf_p, sizeof(float) * tf_resolution,
  // hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy2DToArray(d_tf, 0, 0, tf_p, tf_resolution * sizeof(float),
                                tf_resolution * sizeof(float), 1,
                                hipMemcpyHostToDevice));

  tf.normalized = true;
  tf.filterMode = hipFilterModeLinear;
  tf.addressMode[0] = hipAddressModeBorder;
  tf.addressMode[1] = hipAddressModeBorder;
  // gpuErrchk(hipBindTexture(&offset, tf, d_tf, channelDesc, sizeof(float) *
  // tf_resolution));
  gpuErrchk(hipBindTextureToArray(tf, d_tf, channelDesc));

  return d_tf;
}

__device__
float accumulateOverRay(glm::vec3 rayStart, glm::vec3 step, int nSteps, glm::vec3 worldBounds, bool log=false) {
    float occlusion = 0.0f;
    glm::vec3 pos = rayStart + 3.f*step; // small offset
    for(int i = 0;
        i < nSteps &&           // Stop after max nSteps
        occlusion < 0.99f &&    // Early ray termination
        glm::all(glm::lessThan(pos,worldBounds)) &&           // out of volume
        glm::all(glm::greaterThan(pos,glm::vec3(0.f))); ++i){ // out of volume

      float op = texFetchVolumeWorld(pos, worldBounds, true);
      occlusion += (1-occlusion) * (op);
      // occlusion = max(occlusion, op);
      // if (occlusion > 0.5f) return 1.0f - static_cast<float>(i)/static_cast<float>(nSteps);
      pos += step;
      if (log && (op > 0.f || i > nSteps-5)) {
        printVec("Position", pos);
        printVec("World Bounds", worldBounds);
        printVec("Step", step);
        printVec("Ray Direction", glm::normalize(step));
        printf("Step %03d with Opacity=%1.3f\n\n==============\n", i, occlusion);
        }
    }
    return occlusion;
}

__device__ float sampleOpacityInRayDir(glm::vec3 vox_pos, glm::vec3 offset,
                                       glm::vec3 worldBounds){
    return (1.f * tex1D(tf, texFetchVolumeWorld(vox_pos + 1.f * offset, worldBounds)) +
            0.5f* tex1D(tf, texFetchVolumeWorld(vox_pos + 2.f * offset, worldBounds)) +
            0.33f*tex1D(tf, texFetchVolumeWorld(vox_pos + 3.f * offset, worldBounds)) +
            0.25f*tex1D(tf, texFetchVolumeWorld(vox_pos + 4.f * offset, worldBounds)) +
            0.125f*tex1D(tf, texFetchVolumeWorld(vox_pos + 8.f * offset, worldBounds))) / 2.205f;

}

__global__ void raycast(float *out, glm::ivec3 volSize, glm::vec3 rayDir,
                        float stepSize, glm::vec3 voxelScale, int nSteps) {
  // Get Volume index for this thread
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  int i = 4 * (z * volSize.x * volSize.y + y * volSize.x + x);
  int nVoxel = glm::compMul(volSize);
  if (i >= 4 * nVoxel) {
    printf("Out of bounds with threading idxs");
    return;
  }
  glm::vec3 worldPos = (glm::vec3(x, y, z) + 0.5f) * voxelScale;
  glm::vec3 worldBounds = glm::vec3(volSize) * voxelScale;

  glm::vec3 normal = glm::normalize(-computeGradient(worldPos, worldBounds));
  if (glm::any(glm::isnan(normal))){ normal = glm::vec3(0.f); }
  float weight = glm::clamp(glm::dot(rayDir, normal), 0.f, 1.f);
  float intensity = texFetchVolumeWorld(worldPos, worldBounds);
  float tfd = tex1D(tf, intensity);
  out[i + 0] = normal.x;
  out[i + 1] = intensity;
  out[i + 2] = tfd;

  out[i + 3] += 1.f - accumulateOverRay(worldPos, stepSize * rayDir, nSteps, worldBounds, false);
  // if (weight > 0.f && hasVisibleNeighbors(worldPos / worldBounds, volSize, true)) {
  //   float opacity = accumulateOverRay(worldPos, stepSize * rayDir, nSteps, worldBounds, false);
  //   out[i + 3] += weight * (1 - opacity);
  // }
}

// __device__
// float median(std::vector<float> &v) {
//   size_t idx = v.size() / 2;
//   std::nth_element(v.begin(), v.begin() + idx, v.end());
//   return v[idx];
// }

__device__ float medianBubble(float *v, size_t sz) {
  size_t minValueIndex;
  float bufferData;
  size_t i, j;

  for (j = 0; j <= (sz - 1) / 2; j++) {
    minValueIndex = j;
    for (i = j + 1; i < sz; i++)
      if (v[i] < v[minValueIndex])
        minValueIndex = i;

    bufferData = v[j];
    v[j] = v[minValueIndex];
    v[minValueIndex] = bufferData;
  }

  return v[(sz - 1) / 2];
}

__global__ void replaceWithMedian(float* vol, glm::ivec3 volSize, float value, float *out) {
  size_t X = volSize.x, Y = volSize.y, Z = volSize.z;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  int i = z * X * Y + y * X + x;
  if(vol[i] != value) return;
  glm::ivec3 shifts[26];
  shifts[0]  = glm::ivec3(0, 0, 1);
  shifts[1]  = glm::ivec3(0, 0, -1);
  shifts[2]  = glm::ivec3(0, 1, 0);
  shifts[3]  = glm::ivec3(0, -1, 0);
  shifts[4]  = glm::ivec3(1, 0, 0);
  shifts[5]  = glm::ivec3(-1, 0, 0);
  shifts[6]  = glm::ivec3(0, 1, 1);
  shifts[7]  = glm::ivec3(0, 1, -1);
  shifts[8]  = glm::ivec3(0, -1, 1);
  shifts[9]  = glm::ivec3(0, -1, -1);
  shifts[10] = glm::ivec3(1, 0, 1);
  shifts[11] = glm::ivec3(1, 0, -1);
  shifts[12] = glm::ivec3(-1, 0, 1);
  shifts[13] = glm::ivec3(-1, 0, -1);
  shifts[14] = glm::ivec3(1, 1, 0);
  shifts[15] = glm::ivec3(1, -1, 0);
  shifts[16] = glm::ivec3(-1, 1, 0);
  shifts[17] = glm::ivec3(-1, -1, 0);
  shifts[18] = glm::ivec3(1, 1, 1);
  shifts[19] = glm::ivec3(1, 1, -1);
  shifts[20] = glm::ivec3(1, -1, 1);
  shifts[21] = glm::ivec3(1, -1, -1);
  shifts[22] = glm::ivec3(-1, 1, 1);
  shifts[23] = glm::ivec3(-1, 1, -1);
  shifts[24] = glm::ivec3(-1, -1, 1);
  shifts[25] = glm::ivec3(-1, -1, -1);

  float vec[26];
  size_t count = 0;
  for (size_t i=0; i < 26; ++i) {
    if (0 < x + shifts[i].x && x + shifts[i].x < X &&
        0 < y + shifts[i].y && y + shifts[i].y < Y &&
        0 < z + shifts[i].z && z + shifts[i].z < Z) {
      size_t cur_i = (z + shifts[i].z)*X*Y  +  (y + shifts[i].y)*X  +  (x + shifts[i].x);
      vec[count] = vol[cur_i];
      count++;
    }
  }
  out[i] = medianBubble(vec, count);
}

extern "C" 
#ifdef _WIN32
__declspec(dllexport)
#endif
void shoutout() {
  printf("shoutout()\n");
}

extern "C" 
#ifdef _WIN32
__declspec(dllexport)
#endif
void raycastVolume(float* vol_p, float* tf_p, int* texDims_p, float* voxelScale_p, float* ray_p, float stepsFactor, size_t nRays, float minValue, float* vol_out_p) {
    // printf("raycastVolume()\n");
    glm::ivec3 volSize      = glm::ivec3(texDims_p[0], texDims_p[1], texDims_p[2]);
    size_t     tfResolution = texDims_p[3];
    glm::vec3  voxelScale   = glm::make_vec3(voxelScale_p);
    size_t     nVoxel       = static_cast<size_t>(glm::compMul(volSize));
    float      stepSize     = 0.5f;
    size_t     nSteps       = static_cast<size_t>(ceil(glm::length(glm::vec3(volSize) * voxelScale) * stepsFactor / stepSize));
    // Metadata
    printf("Raycasting Volume (%d, %d, %d, Total: %d). TF has res of %d.\n", volSize.x, volSize.y, volSize.z, nVoxel, tfResolution);
    printf("Doing %d steps of length %1.4f for a total ray length of %1.2f\n", nSteps, stepSize, nSteps * stepSize);


    // float4* vol_out_f4p = (float4 *) vol_out_p;
    // Allocate result memory on gpu
    float* d_vol_out;
    gpuErrchk(hipMalloc((void **) &d_vol_out, nVoxel * 4 * sizeof(float)));
    gpuErrchk(hipMemcpy(d_vol_out, vol_out_p, nVoxel * 4 * sizeof(float), hipMemcpyHostToDevice));

    dim3 grid(    1,     volSize.y, volSize.z);
    dim3 block(volSize.x,    1,          1   );
    // Replace invalid voxels (==vol.min()) with median of neighborhood
    // float *d_vol, *d_vol_r;
    // gpuErrchk(hipMalloc((void **) &d_vol,   nVoxel * sizeof(float)));
    // gpuErrchk(hipMalloc((void **) &d_vol_r, nVoxel * sizeof(float)));
    // gpuErrchk(hipMemcpy(d_vol, vol_p,       nVoxel * sizeof(float), hipMemcpyHostToDevice));
    // replaceWithMedian<<<grid, block>>>(d_vol, volSize, minValue, d_vol_r);
    // gpuErrchk(hipDeviceSynchronize());
    // gpuErrchk(hipMemcpy(vol_p, d_vol_r, nVoxel * sizeof(float), hipMemcpyDeviceToHost));
    // Upload volume as texture
    auto d_vol_tex = uploadVolume(vol_p, volSize);
    auto d_tf_tex = uploadTransferFunction(tf_p, tfResolution);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipPeekAtLastError());
    // printf("Starting now...\n");
    for (size_t i = 0; i < 3*nRays; i+=3){
      glm::vec3 rayDir = glm::vec3(ray_p[i], ray_p[i + 1], ray_p[i + 2]);
      // glm::vec3 rayDir = glm::normalize(glm::vec3(ray_p[i], ray_p[i + 1], ray_p[i + 2]) / voxelScale);
      raycast<<<grid, block>>>(d_vol_out, volSize, rayDir, stepSize, voxelScale, nSteps);
      // if(rayDir.z > 0.f) { printf("!!!!!!!!!!! we got z > 0"); printVec("rayDir", rayDir); }
      gpuErrchk( hipDeviceSynchronize() );
      // printf("Rays cast: %d/%d (%02.2f%)\r", i/3, nRays, 100.f*(i/3.0f)/nRays);
      // std::cout.flush();
    }
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk( hipMemcpy(vol_out_p, d_vol_out, nVoxel * 4 * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk( hipFree(d_vol_out));
    gpuErrchk( hipFreeArray(d_tf_tex));
    gpuErrchk( hipFreeArray(d_vol_tex));
    gpuErrchk( hipUnbindTexture(tex));
    gpuErrchk( hipUnbindTexture(tf));
}
